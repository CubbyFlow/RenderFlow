#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017 NVIDIA Corporation
// SPDX-License-Identifier: Apache-2.0
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//-----------------------------------------------------------------------------

#include "optix_extra_math.cuh"

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertex_buffer;     
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   vindex_buffer;    // position indices 


rtBuffer<int3>   nindex_buffer;    // normal indices
rtBuffer<int3>   tindex_buffer;    // texcoord indices
rtBuffer<uint>   mindex_buffer;    // per-face material index

rtDeclareVariable(float3, back_hit_point,	attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,	attribute front_hit_point, ); 
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,	attribute shading_normal, ); 
rtDeclareVariable(float4, deep_color,		attribute deep_color, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void mesh_intersect( int primIdx )
{
  int3 v_idx = vindex_buffer[primIdx];

  float3 p0 = vertex_buffer[ v_idx.x ];
  float3 p1 = vertex_buffer[ v_idx.y ];
  float3 p2 = vertex_buffer[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      // Calculate normals and tex coords 
      float3 geo_n = normalize( n );
      int3 n_idx = nindex_buffer[ primIdx ];
      
	  shading_normal = geo_n;
	  
	  // INTERPOLATED NORMALS
	  if ( normal_buffer.size() == 0 || n_idx.x < 0 || n_idx.y < 0 || n_idx.z < 0 ) {
        shading_normal = geo_n;
      } else {
        float3 n0 = normal_buffer[ n_idx.x ];
        float3 n1 = normal_buffer[ n_idx.y ];
        float3 n2 = normal_buffer[ n_idx.z ];
        shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
      }

      geometric_normal = geo_n;

      int3 t_idx = tindex_buffer[ primIdx ];
      if ( texcoord_buffer.size() == 0 || t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0 ) {
        texcoord = make_float3( 0.0f, 0.0f, 0.0f );
      } else {

        float2 t0 = texcoord_buffer[ t_idx.x ];
        float2 t1 = texcoord_buffer[ t_idx.y ];
        float2 t2 = texcoord_buffer[ t_idx.z ];
        texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
      }

      refine_and_offset_hitpoint( ray.origin + t*ray.direction, ray.direction,
                                  geo_n, p0,
                                  back_hit_point, front_hit_point );

	  deep_color = make_float4 ( 0, 0, 0, 1 );

      rtReportIntersection( mindex_buffer[primIdx] );
    }
  }
}


RT_PROGRAM void mesh_bounds (int primIdx, float result[6])
{
  const int3 v_idx = vindex_buffer[primIdx];

  const float3 v0 = vertex_buffer[ v_idx.x ];
  const float3 v1 = vertex_buffer[ v_idx.y ];
  const float3 v2 = vertex_buffer[ v_idx.z ];
  const float  area = length(cross(v1-v0, v2-v0));

  optix::Aabb* aabb = (optix::Aabb*)result;

  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}


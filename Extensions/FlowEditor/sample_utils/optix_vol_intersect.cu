#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017 NVIDIA Corporation
// SPDX-License-Identifier: Apache-2.0
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//-----------------------------------------------------------------------------


#include "optix_gvdb.cuh"

rtBuffer<float3>		  brick_buffer;

rtDeclareVariable(uint,	  mat_id, , );
rtDeclareVariable(float3, light_pos, , );

rtDeclareVariable(float3, back_hit_point,	attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,	attribute front_hit_point, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,	attribute shading_normal, ); 
rtDeclareVariable(float4, deep_color,		attribute deep_color, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

struct RayInfo
{
	float3	result;
	float	length; 
	float	alpha;
	int		depth;
	int		rtype;
};
rtDeclareVariable( RayInfo, ray_info, rtPayload, );

//------ Intersection Program

RT_PROGRAM void vol_intersect( int primIdx )
{
	float3 hit = make_float3(NOHIT,NOHIT,NOHIT);	
	float3 norm = make_float3(0,0,0);	
	float4 hclr = make_float4(1,1,1,1);
	float t;

	//-- Ray march
	// If the Optix transform node has been set up correctly, then the ray is in GVDB's coordinate system.
	const float3 orig = ray.origin;
	const float3 dir = ray.direction;
	rayCast(&gvdbObj, gvdbChan, orig, dir, hit, norm, hclr, raySurfaceTrilinearBrick);
	if ( hit.z == NOHIT) return;	
	t = length ( hit - ray.origin );

	// report intersection to optix
	if ( rtPotentialIntersection( t ) ) {	

		shading_normal = norm;		
		geometric_normal = norm;
		// Transform from GVDB's coordinate space to OptiX's coordinate space
		front_hit_point = hit + shading_normal * 2;
		back_hit_point = hit - shading_normal * 4;
		deep_color = hclr;
		//if ( ray_info.rtype == SHADOW_RAY ) deep_color.w = (hit.x!=NOHIT) ? 0 : 1;

		rtReportIntersection( mat_id );
	}
}

RT_PROGRAM void vol_deep( int primIdx )
{
	float3 hit = make_float3(0,0,NOHIT);	
	float3 norm = make_float3(0,1,0);
	float4 clr = make_float4(0,0,0,1);	
	if (ray_info.rtype == MESH_RAY ) return;

	float3 orig = ray.origin;
	float3 dir = ray.direction;

	// ---- Debugging
	// Uncomment this code to demonstrate tracing of the bounding box 
	// surrounding the volume.
	/* hit = rayBoxIntersect ( orig, dir, gvdbObj.bmin, gvdbObj.bmax );
	if ( hit.z == NOHIT ) return;
	const float t2 = hit.x;
	if ( rtPotentialIntersection ( t2 ) ) {
		shading_normal = norm;		
		geometric_normal = norm;
		front_hit_point = orig + hit.x * dir;
		back_hit_point  = orig + hit.y * dir;
		deep_color = make_float4( front_hit_point/200.0, 0.5);	
		rtReportIntersection( 0 );
	}
	return; */

	//-- Raycast		
	rayCast ( &gvdbObj, gvdbChan, orig, dir, hit, norm, clr, rayDeepBrick );	
	if ( hit.x==0 && hit.y == 0) return;

	// Note that rayDeepBrick sets hit.x and hit.y to the front and back brick intersection points in GVDB's coordinate
	// system, in contrast to the other functions in this file.
	const float t = hit.x;

	if ( rtPotentialIntersection( t ) ) {

		shading_normal = norm;		
		geometric_normal = norm;
		// Transform from GVDB's coordinate space to the application's coordinate space
		front_hit_point = orig + hit.x * dir;
		back_hit_point  = orig + hit.y * dir;
		deep_color = make_float4 ( fxyz(clr), 1.0-clr.w );		

		rtReportIntersection( 0 );			
	}
}

RT_PROGRAM void vol_levelset ( int primIdx )
{
	float3 hit = make_float3(NOHIT,NOHIT,NOHIT);	
	float3 norm = make_float3(0,0,0);
	float4 hclr = make_float4(0,1,0,1);	
	float t;
	
	//if (ray_info.rtype == SHADOW_RAY && ray_info.depth >= 1) return;

	// Transform from application space to GVDB's coordinate space
	float3 orig = ray.origin;
	float3 dir = ray.direction;

	//-- Ray march		
	if (ray_info.rtype == REFRACT_RAY) {		
		if (ray_info.depth == 2) return;
		rayCast(&gvdbObj, gvdbChan, orig, dir, hit, norm, hclr, raySurfaceTrilinearBrick);
	}	else {
		rayCast(&gvdbObj, gvdbChan, orig, dir, hit, norm, hclr, rayLevelSetBrick);
	}
	if ( hit.z == NOHIT) return;

	// Transform from GVDB's coordinate space to application space
	t = length ( hit - ray.origin );

	// report intersection to optix
	if ( rtPotentialIntersection( t ) ) {	

		shading_normal = norm;		
		geometric_normal = norm;
		front_hit_point = hit;
		back_hit_point = hit - shading_normal * .2;
		deep_color = hclr;

		rtReportIntersection( mat_id );
	}
}


RT_PROGRAM void vol_bounds (int primIdx, float result[6])
{
	// AABB bounds is just the brick extents	
	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = brick_buffer[ primIdx*2 ];
	aabb->m_max = brick_buffer[ primIdx*2+1 ];
}


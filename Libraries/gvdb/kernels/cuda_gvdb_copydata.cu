#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017 NVIDIA Corporation
// SPDX-License-Identifier: Apache-2.0
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//-----------------------------------------------------------------------------
// File: cuda_gvdb_copydata.cu
//
// GVDB Data Transfers
// - CopyData		3D volume into sub-volume
// - CopyDataZYX	3D volume into sub-volume with ZYX swizzle
// - RetreiveData	3D sub-volume into cuda buffer
// - CopyTexToBuf	2D texture into cuda buffer
// - CopyBufToTex	cuda buffer into 2D texture
//-----------------------------------------------

#include "cuda_math.cuh"

// Zero memory of 3D volume
extern "C" __global__ void kernelFillTex ( int3 res, int dsize, hipSurfaceObject_t volTexOut )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;

	surf3Dwrite ( 0, volTexOut, t.x*dsize, t.y, t.z );
}

// Copy 3D texture into sub-volume of another 3D texture (char)
extern "C" __global__ void kernelCopyTexC ( int3 offs, int3 res, hipSurfaceObject_t volTexOut )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;
	uchar val = surf3Dread<uchar>(volTexOut, t.x * sizeof(uchar), t.y, t.z);
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(uchar), (t.y+offs.y), (t.z+offs.z) );
}

// Copy 3D texture into sub-volume of another 3D texture (float)
extern "C" __global__ void kernelCopyTexF ( int3 offs, int3 res, hipSurfaceObject_t volTexOut )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;	
	float val = surf3Dread<float>(volTexOut, t.x * sizeof(float), t.y, t.z);
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(float), (t.y+offs.y), (t.z+offs.z) );
}

// Copy linear memory as 3D volume into sub-volume of a 3D texture
extern "C" __global__ void kernelCopyBufToTexC ( int3 offs, int3 res, uchar* inbuf, hipSurfaceObject_t volTexOut)
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;	
	unsigned char val = inbuf[ (t.z*res.y + t.y)*res.x + t.x ];	
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(uchar), (t.y+offs.y), (t.z+offs.z) );
}
// Copy linear memory as 3D volume into sub-volume of a 3D texture
extern "C" __global__ void kernelCopyBufToTexF ( int3 offs, int3 res, float* inbuf, hipSurfaceObject_t volTexOut)
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;	
	float val = inbuf[ (t.z*res.y + t.y)*res.x + t.x ];	
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(float), (t.y+offs.y), (t.z+offs.z) );
}

// Copy 3D texture into sub-volume of another 3D texture with ZYX swizzle (float)
extern "C" __global__ void kernelCopyTexZYX (  int3 offs, int3 res, hipSurfaceObject_t volTexInF, hipSurfaceObject_t volTexOut )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= res.x || t.y >= res.y || t.z >= res.z ) return;
	float val = surf3Dread<float>(volTexInF, t.z * sizeof(float), t.y, t.x);
	surf3Dwrite ( val, volTexOut, (t.x+offs.x)*sizeof(float), (t.y+offs.y), (t.z+offs.z) );
}

// Retrieve 3D texture into linear memory (float)
extern "C" __global__ void kernelRetrieveTexXYZ ( int3 offs, int3 brickRes, float* buf, hipSurfaceObject_t volTexInF )
{
	uint3 t = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;	
	if ( t.x >= brickRes.x || t.y >= brickRes.y || t.z >= brickRes.z ) return;
	float val = surf3Dread<float>(volTexInF, (t.x + offs.x) * sizeof(float), t.y + offs.y, t.z + offs.z);
	buf[ (t.x*brickRes.y + t.y)*brickRes.x + t.z ] = val;
}